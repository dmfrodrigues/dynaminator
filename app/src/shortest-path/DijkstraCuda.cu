#include "shortest-path/DijkstraCuda.hpp"

#include <algorithm>
#include <cassert>
#include <chrono>
#include <iostream>
#include <queue>
#include <utility>

#include "structs/BinaryHeap.hpp"

using namespace std;

typedef Graph::Node Node;
typedef Graph::Edge::Weight Weight;
typedef Graph::Edge Edge;
template <class K, class V>
using umap = std::unordered_map<K, V>;
typedef umap<Node, Weight> dist_t;
typedef umap<Node, Node> prev_t;
typedef BinaryHeap<std::pair<Weight, Node>> MinPriorityQueue;
typedef std::chrono::high_resolution_clock hrc;

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        cerr << "GPUassert: " << hipGetErrorString(code) << " (" << file << ":" << line << ")" << endl;
        if (abort) exit(code);
    }
}

void DijkstraCuda::initialize(const Graph *G, const list<Node> &s) {
    const vector<Node> &nodes = G->getNodes();

    numberNodes = (nodes.empty() ? 1 : *max_element(nodes.begin(), nodes.end()) + 1);
    gpuErrchk(hipMallocManaged(&adj, numberNodes * sizeof(pair<Edge::ID, Edge::ID>)));

    numberEdges = 0;
    for (const Node &u : nodes) numberEdges += G->getAdj(u).size();
    gpuErrchk(hipMallocManaged(&edges, numberEdges * sizeof(Edge)));

    size_t edgeIdx = 0;
    for (const Node &u : nodes) {
        const auto &es = G->getAdj(u);
        adj[u] = pair<Edge::ID, Edge::ID>(edgeIdx, edgeIdx + es.size());
        copy(es.begin(), es.end(), &edges[edgeIdx]);
        edgeIdx += es.size();
    }
    assert(edgeIdx == numberEdges);

    numberStartNodes = s.size();
    gpuErrchk(hipMallocManaged(&startNodes, numberStartNodes * sizeof(Node)));
    copy(s.begin(), s.end(), startNodes);

    gpuErrchk(hipMallocManaged(&prev, numberNodes * sizeof(Edge *)));
    gpuErrchk(hipMallocManaged(&dist, numberNodes * sizeof(Weight *)));
    fill(prev, prev + numberNodes, nullptr);
    fill(dist, dist + numberNodes, nullptr);
    for (const Node &u : s) {
        gpuErrchk(hipMallocManaged(&prev[u], numberNodes * sizeof(Edge)));
        gpuErrchk(hipMallocManaged(&dist[u], numberNodes * sizeof(Weight)));
        fill(prev[u], prev[u] + numberNodes, Graph::EDGE_INVALID);
        fill(dist[u], dist[u] + numberNodes, Edge::WEIGHT_INF);
    }
}

void runDijkstra(
    size_t numberNodes, size_t numberEdges,
    const Edge *edges, const pair<Edge::ID, Edge::ID> *adj,
    Node s,
    Edge *prev,
    Weight *dist) {
    vector<MinPriorityQueue::Element *> elements(numberNodes);

    MinPriorityQueue Q;
    Q.reserve(numberNodes);

    dist[s] = 0;
    elements[s] = &Q.push({0, s});
    while (!Q.empty()) {
        Node u = Q.top().second;
        Q.pop();
        for (size_t i = adj[u].first; i < adj[u].second; ++i) {
            const Edge &e = edges[i];
            Weight c_ = dist[u] + e.w;
            Weight &distV = dist[e.v];
            if (c_ < distV) {
                if (elements[e.v])
                    elements[e.v]->decreaseKey({c_, e.v});
                else
                    elements[e.v] = &Q.push({c_, e.v});
                distV = c_;
                prev[e.v] = e;
            }
        }
    }
}

void DijkstraCuda::run() {
    for (size_t i = 0; i < numberStartNodes; ++i) {
        const Node &s = startNodes[i];
        runDijkstra(numberNodes, numberEdges, edges, adj, s, prev[s], dist[s]);
    }
}

Edge DijkstraCuda::getPrev(Node s, Node d) const {
    if (s >= numberNodes || prev[s] == nullptr)
        throw out_of_range("s is not a valid start node");
    if (d >= numberNodes)
        throw out_of_range("d is not a valid destination node");
    return prev[s][d];
}

Weight DijkstraCuda::getPathWeight(Node s, Node d) const {
    if (s >= numberNodes || dist[s] == nullptr)
        throw out_of_range("s is not a valid start node");
    if (d >= numberNodes)
        throw out_of_range("d is not a valid destination node");
    return dist[s][d];
}

bool DijkstraCuda::hasVisited(Node s, Node u) const {
    return getPathWeight(s, u) != Edge::WEIGHT_INF;
}
